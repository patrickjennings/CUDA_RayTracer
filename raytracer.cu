#include "hip/hip_runtime.h"
/*
 * Modified by:
 * Patrick Jennings
 * Stevie Frederick
 *
 */

#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <math_functions.h>
#include <cutil_math.h>

#define STOCHASTIC 0

texture<float4, 1, hipReadModeElementType> triangle_texture; // the scene triangles store in a 1D float4 texture 
                                                              // the triangles is stored as the 3 vertices's after each other
__device__ inline int absint(int x) {
	return (x > 0) ? x : -x;
}


// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
	r = clamp(r, 0.0f, 255.0f);
	g = clamp(g, 0.0f, 255.0f);
	b = clamp(b, 0.0f, 255.0f);
	return (int(r)<<16) | (int(g)<<8) | int(b); // notice switch red and blue to counter the GL_BGRA
}

// Ray structure
struct Ray
{	
	__device__ Ray(){};
	__device__ Ray(const float3 &o,const float3 &d)
	{
		ori = o;
		dir = d;
		dir = normalize(dir);
		inv_dir = make_float3(1.0/dir.x,1.0/dir.y,1.0/dir.z);
	}
	float3 ori;
	float3 dir;
	float3 inv_dir;
};

struct HitRecord
{
	__device__ HitRecord() {t = UINT_MAX;hit_index = -1; color = make_float3(0,0,0);}
	__device__ void resetT(){t = UINT_MAX; hit_index = -1;}
	float t;
	float3 color;
	float3 normal;
	int hit_index; 
	
};

// intersection code --------------------------------------------------
__device__ int RayBoxIntersection(const float3 &BBMin, const float3 &BBMax, const float3 &RayOrg, const float3 &RayDirInv, float &tmin, float &tmax)
{
	float l1   = (BBMin.x - RayOrg.x) * RayDirInv.x;
	float l2   = (BBMax.x - RayOrg.x) * RayDirInv.x;
	tmin = fminf(l1,l2);
	tmax = fmaxf(l1,l2);

	l1   = (BBMin.y - RayOrg.y) * RayDirInv.y;
	l2   = (BBMax.y - RayOrg.y) * RayDirInv.y;
	tmin = fmaxf(fminf(l1,l2), tmin);
	tmax = fminf(fmaxf(l1,l2), tmax);

	l1   = (BBMin.z - RayOrg.z) * RayDirInv.z;
	l2   = (BBMax.z - RayOrg.z) * RayDirInv.z;
	tmin = fmaxf(fminf(l1,l2), tmin);
	tmax = fminf(fmaxf(l1,l2), tmax);

	return ((tmax >= tmin) && (tmax >= 0.0f));
}

// the classic ray triangle intersection: http://www.cs.virginia.edu/~gfx/Courses/2003/ImageSynthesis/papers/Acceleration/Fast%20MinimumStorage%20RayTriangle%20Intersection.pdf
__device__ float RayTriangleIntersection(const Ray &r, 
										 const float3 &v0, 
										 const float3 &edge1, 
										 const float3 &edge2)
{  

	float3 tvec = r.ori- v0;  
	float3 pvec = cross(r.dir, edge2);  
	float  det  = dot(edge1, pvec);  

	det = __fdividef(1.0f, det);  

	float u = dot(tvec, pvec) * det;  

	if (u < 0.0f || u > 1.0f)  
		return -1.0f;  

	float3 qvec = cross(tvec, edge1);  

	float v = dot(r.dir, qvec) * det;  

	if (v < 0.0f || (u + v) > 1.0f)  
		return -1.0f;  

	return dot(edge2, qvec) * det;  
}  

__device__ int RaySphereIntersection(const Ray  &ray, const float3 sphere_center, const float sphere_radius, float &t)
{
	float b, c, d;

	float3 sr = ray.ori - sphere_center;
	b =  dot(sr,ray.dir);
	c = dot(sr,sr) - (sphere_radius*sphere_radius);
	d = b*b - c;
	if (d > 0) 
	{
		float e = sqrt(d);
		float t0 = -b-e;
		if(t0 < 0)
			t = -b+e;
		else
			t = min(-b-e,-b+e);
		return 1;
	}
	return 0;
}

__global__ void raytrace( unsigned int *out_data,
						   const int w,
						   const int h,
						   const int number_of_triangles,
						   const float3 a, const float3 b, const float3 c, 
						   const float3 campos,
						   const float3 light_pos,
						   const float3 light_color,
						   const float3 scene_aabb_min, 
						   const float3 scene_aabb_max)
{

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	float3 val = make_float3(0,0,0);
	int num_rays = 8;
	for(int p = 1; p <= num_rays; p++) {
	float step;
	if(STOCHASTIC)
		step = (float)1 / (float)(out_data[y * w + x]%50 + 5);		// Pseudorandom step size.
	else
		step = 1.0f / (float)p;
	float xf = (x-step)/((float)w);
	float yf = (y-step)/((float)h);

	int ray_depth = 0;
	bool continue_path = true;

	float3 t1 = c+(a*xf);
	float3 t2 = b*yf;
	float3 image_pos = t1 + t2;
	Ray r(image_pos,image_pos-campos);
	HitRecord hit_r;

	float t_min,t_max;
	continue_path = RayBoxIntersection(scene_aabb_min, scene_aabb_max, r.ori, r.inv_dir,t_min, t_max);
	hit_r.color = make_float3(0,0,0);

	// hack to display the light source we simple make a ray sphere intersection and 
	// compare the depth with the found t value from the triangles
	float sphere_t;
	bool sphere_hit = RaySphereIntersection(r,light_pos,2.0,sphere_t);
	
	if(sphere_hit && sphere_t > 0.001)
	{
		if(!continue_path)
		{
			hit_r.color = light_color;
		}
		sphere_hit = true;
	}

	while(continue_path && ray_depth < 4)
	{
		
		// search through the triangles and find the nearest hit point
		for(int i = 0; i < number_of_triangles; i++)
		{
			float4 v0 = tex1Dfetch(triangle_texture,i*3);
			float4 e1 = tex1Dfetch(triangle_texture,i*3+1);
			float4 e2 = tex1Dfetch(triangle_texture,i*3+2);

			float t = RayTriangleIntersection(r, make_float3(v0.x,v0.y,v0.z),make_float3(e1.x,e1.y,e1.z), make_float3(e2.x,e2.y,e2.z));

			if(t < hit_r.t && t > 0.001)
			{
				hit_r.t = t; 
				hit_r.hit_index = i;
			}
		}

		if(sphere_hit && sphere_t < hit_r.t)
		{
			hit_r.color += light_color;
			continue_path = false;
			break;
		}

		if(hit_r.hit_index >= 0)
		{
			ray_depth++;
			
			// create the normal
			float4 e1 = tex1Dfetch(triangle_texture,hit_r.hit_index*3+1);
			float4 e2 = tex1Dfetch(triangle_texture,hit_r.hit_index*3+2);

			hit_r.normal = cross(make_float3(e1.x,e1.y,e1.z), make_float3(e2.x,e2.y,e2.z));
			hit_r.normal = normalize(hit_r.normal);

			// calculate simple diffuse light
			float3 hitpoint = r.ori + r.dir *hit_r.t;
			float3 L = light_pos - hitpoint;
			float dist_to_light = length(L);
			
			L = normalize(L);
			float diffuse_light = max( dot(L,hit_r.normal), 0.0);
			diffuse_light = min( (diffuse_light),1.0);
			//calculate simple specular light
			float3 H = L + (-r.dir);
			H = normalize(H);
			float specular_light = powf(max(dot(H,hit_r.normal),0.0),25.0f);

			diffuse_light  *=  16.0/dist_to_light;
			specular_light *=  16.0/dist_to_light;

			clamp(diffuse_light, 0.0f, 1.0f);
			clamp(specular_light, 0.0f, 1.0f);

			hit_r.color += light_color * diffuse_light + make_float3(1.0,1.0,1.0)*specular_light*0.2 + make_float3(0.2,0.2,0.2);
			
			// create a shadow ray
			Ray shadow_ray(hitpoint, L);
			for(int i = 0; i < number_of_triangles; i++)
			{
				float4 v0 = tex1Dfetch(triangle_texture,i*3);
				float4 e1 = tex1Dfetch(triangle_texture,i*3+1);
				float4 e2 = tex1Dfetch(triangle_texture,i*3+2);
				float t = RayTriangleIntersection(shadow_ray, make_float3(v0.x,v0.y,v0.z),make_float3(e1.x,e1.y,e1.z), make_float3(e2.x,e2.y,e2.z));

				if(t > 0.025) // there is a blocker on the path to the light
				{
					hit_r.color *= 0.25;
					break;
				}
			}

			if(e1.w > 0) // this is also a little hack to include a specular material
			{
				hit_r.resetT();
				r = Ray(hitpoint, reflect(r.dir,hit_r.normal));
			}
			else
			{
				continue_path = false;
			}
		}
		else
		{
			continue_path = false;
			hit_r.color += make_float3(0.5,0.5,0.95*yf+0.3);
		}
	}

	if(ray_depth >= 1 || sphere_hit)
	{
		ray_depth = max(ray_depth,1);
		hit_r.color /= ray_depth; // normalize the colors
	}
	else
	{
		hit_r.color = make_float3(0.5,0.5,yf+0.3);
	}
	
	if(absint(val.x*255/p-hit_r.color.x*255) < 25 && absint(val.y*255/p-hit_r.color.y*255) < 25
			&& absint(val.z*255/p-hit_r.color.z*255) < 25) {
		val += hit_r.color;                             // Add the current ray color to the total pixel color
		num_rays = p;
		break;
	}
	else
		val += hit_r.color;                             // Add the current ray color to the total pixel color
	}
	val /= num_rays;				// Normalize the pixel color.
	out_data[y * w + x] = rgbToInt(val.x*255,val.y*255,val.z*255);	// Set the pixel color.
}

extern "C" 
{
	void RayTraceImage(unsigned int *pbo_out, int w, int h, int number_of_triangles,
		               float3 a, float3 b, float3 c, 
		               float3 campos,
					   float3 light_pos,
					   float3 light_color,
					   float3 scene_aabbox_min, float3 scene_aabbox_max)
	{

		dim3 block(10,10,1);
		dim3 grid(w/block.x,h/block.y, 1);
		raytrace<<<grid, block>>>(pbo_out,w,h,number_of_triangles,a,b,c,campos,light_pos,light_color,scene_aabbox_min,scene_aabbox_max);

	}

	void bindTriangles(float *dev_triangle_p, unsigned int number_of_triangles)
	{
		triangle_texture.normalized = false;                      // access with normalized texture coordinates
		triangle_texture.filterMode = hipFilterModePoint;        // Point mode, so no 
		triangle_texture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(float4)*number_of_triangles*3;       
		hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
		hipBindTexture(0,triangle_texture,dev_triangle_p,channelDesc,size);
	}
}
